#include "hip/hip_runtime.h"
#include "fitzhugh_1961.h"
#include <stddef.h>
#include <stdint.h>
#include "model_gpu_utils.h"

extern "C" SET_ODE_INITIAL_CONDITIONS_GPU(set_model_initial_conditions_gpu) 
{

    print_to_stdout_and_file("Using fitzhugh_1961 GPU model\n");

    // execution configuration
    const int GRID  = (num_volumes + BLOCK_SIZE - 1)/BLOCK_SIZE;

    size_t size = num_volumes*sizeof(real);

    check_cuda_error(hipMallocPitch((void **) &(*sv), &pitch_h, size, (size_t )NEQ));
    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(pitch), &pitch_h, sizeof(size_t)));


    kernel_set_model_inital_conditions <<<GRID, BLOCK_SIZE>>>(*sv, num_volumes);

    check_cuda_error( hipPeekAtLastError() );
    hipDeviceSynchronize();
    return pitch_h;

}

extern "C" SOLVE_MODEL_ODES_GPU(solve_model_odes_gpu) {

    // execution configuration
    const int GRID  = ((int)num_cells_to_solve + BLOCK_SIZE - 1)/BLOCK_SIZE;


    size_t stim_currents_size = sizeof(real)*num_cells_to_solve;
    size_t cells_to_solve_size = sizeof(uint32_t)*num_cells_to_solve;

    real *stims_currents_device;
    check_cuda_error(hipMalloc((void **) &stims_currents_device, stim_currents_size));
    check_cuda_error(hipMemcpy(stims_currents_device, stim_currents, stim_currents_size, hipMemcpyHostToDevice));


    //the array cells to solve is passed when we are using and adapative mesh
    uint32_t *cells_to_solve_device = NULL;
    if(cells_to_solve != NULL) {
        check_cuda_error(hipMalloc((void **) &cells_to_solve_device, cells_to_solve_size));
        check_cuda_error(hipMemcpy(cells_to_solve_device, cells_to_solve, cells_to_solve_size, hipMemcpyHostToDevice));
    }
    solve_gpu <<<GRID, BLOCK_SIZE>>>(dt, sv, stims_currents_device, cells_to_solve_device, num_cells_to_solve, num_steps);

    check_cuda_error( hipPeekAtLastError() );

    check_cuda_error(hipFree(stims_currents_device));
    if(cells_to_solve_device) check_cuda_error(hipFree(cells_to_solve_device));

}

__global__ void kernel_set_model_inital_conditions(real *sv, int num_volumes) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadID < num_volumes) {

         *((real * )((char *) sv + pitch * 0) + threadID) = 0.000000f; //V millivolt 
         *((real * )((char *) sv + pitch * 1) + threadID) = 0.000000f; //h dimensionless 
    }
}

// Solving the model for each cell in the tissue matrix ni x nj
__global__ void solve_gpu(real dt, real *sv, real* stim_currents,
                          uint32_t *cells_to_solve, uint32_t num_cells_to_solve,
                          int num_steps)
{
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        real rDY[NEQ];

        for (int n = 0; n < num_steps; ++n) {

            RHS_gpu(sv, rDY, stim_currents[threadID], sv_id);

            for(int i = 0; i < NEQ; i++) {
                *((real *) ((char *) sv + pitch * i) + sv_id) = dt * rDY[i] + *((real *) ((char *) sv + pitch * i) + sv_id);
            }            

        }

    }
}

inline __device__ void RHS_gpu(real *sv_, real *rDY_, real stim_current, int threadID_) {

    //State variables
    const real V_old_ =  *((real*)((char*)sv_ + pitch * 0) + threadID_);
    const real h_old_ =  *((real*)((char*)sv_ + pitch * 1) + threadID_);

    //Parameters
    const real alpha = -0.100000000000000e+00f;
    const real gamma = 3.000000000000000e+00f;
    const real epsilon = 5.000000000000000e-03f;

    real calc_I_stim = stim_current;

    rDY_[0] = (( V_old_*(V_old_ - alpha)*(1.00000 - V_old_) - h_old_) + calc_I_stim);
    rDY_[1] = epsilon*(V_old_ -  gamma*h_old_);

}
